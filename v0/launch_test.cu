#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <cstdlib> // for rand() and srand()
#include <ctime> // for time()
#include <stdexcept>
#include <vector>
#include "kmeans_base.h"
#include "kmeans_base_gpu.cuh"
#include "error.cuh"

using namespace std;


void readCoordinate(float *data, int *label, const int n_features, int &n) {
    std::ifstream ifs;
    ifs.open("/home/gg/Desktop/kmeans/data/test.csv", std::ios::in);
    if (ifs.fail()) {
        std::cout << "No such file or directory: sample_1e6_fea_100_class_4_lable_1_ninfo_8.csv" << std::endl;
        exit(1);
    }
    std::string line;
    while (std::getline(ifs, line)) {
        std::stringstream sstream(line);
        if (line.empty()) continue;
        int m = 0;
        std::string s_fea;
        while (std::getline(sstream, s_fea, ',')) {
            if (m < n_features) data[n * n_features + m] = std::stod(s_fea);
            else label[n] = std::stoi(s_fea);
            m++;
        }
        n++;
    }
    ifs.close();
}

template <typename T>
void printVecInVec(const T* vecInVec, int rows, int cols, const std::string& title) {
    std::cout << title << ":" << std::endl;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << vecInVec[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

void timing(
    float *data, 
    int *label, 
    float *clusters, 
    const int numClusters, 
    const int n_features, 
    const int n_samples,
    const int method) {
    
    Kmeans *model;
    switch (method)
    {
    case 0:
        model = new Kmeans(numClusters, n_features, clusters, n_samples, 50, 1e-5);
        break;
    case 1:
        model = new KmeansGPU(numClusters, n_features, clusters, n_samples, 50, 1e-5);
        break; 
    default:
        break;
    }

    std::cout << "*********starting fitting*********" << std::endl;

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    model->fit(data);

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    
    printf("Time = %g ms.\n", elapsedTime);

    std::cout << "********* final clusters**********" << std::endl;
    printVecInVec<float>(model->m_clusters, 4, 4, "clusters");
    std::cout << "*********    accuracy  **********" << std::endl;
    std::cout << "model accuracy : " << model->accuracy(label) << std::endl;
    printVecInVec<int>(model->m_sampleClasses, 1, 10, "sampleClasses_10");

    delete model;
}


int main() {
    int N = 0; // 实际读取的样本数量
    const int n_nums = 100; // 数据中，有100个样本
    const int n_features = 4; // 每个样本有4个特征
    const int n_classes = 4;
    float data[n_features * n_nums]; // 需要 4 * 100 的数组容量来存储数据
    int label[n_nums]; // 对应的标签数组
    readCoordinate(data, label, n_features, N);
    std::cout << "num of samples : " << N << std::endl;

    // 数据初始化
    
    int cidx[4] = {0};
    srand(time(NULL));
    for(int i = 0; i < n_classes; i++) cidx[i] = rand() % 100;
    float clusters[n_classes * n_features] = {0};
    for(int i = 0; i < n_classes; i++){
        for(int j = 0; j < n_features; j++){
            clusters[i * n_features + j] = data[cidx[i] * n_features + j];
        }
    }

    printVecInVec<float>(clusters, 4, 4, "clusters");



    std::cout << "Using CPU:" << std::endl;
    timing(data, label, clusters, n_classes, n_features, N, 0);


    std::cout << "Using CUDA:" << std::endl;
    timing(data, label, clusters, n_classes, n_features, N, 1);



    return 0;
}